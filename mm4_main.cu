
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#define threshold 0.0001
#define BLOCK_SIZE 16

void checkCUDAError(const char *msg);

hipEvent_t start, stop;
float tstart, elapsedTime;

__global__ void ab_gpu(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void ab_gpu_small_square(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void ab_gpu_vertical(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk, int Ni_new, int Nj_new);
__global__ void ab_gpu_horizontal(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk, int Ni_new, int Nj_new);

__global__ void abT_gpu(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void aTb_gpu(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);
__global__ void aTbT_gpu(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk);

void ab_seq(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
  int i, j, k;

  for (i = 0; i < Ni; i++)
   for (j = 0; j < Nj; j++)
    for (k = 0; k < Nk; k++)
// C[i][j] = C[i][j] + A[i][k]*B[k][j];
     C[i*Nj+j]=C[i*Nj+j]+A[i*Nk+k]*B[k*Nj+j];
}

void abT_seq(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
  int i, j, k;

  for (i = 0; i < Ni; i++)
   for (j = 0; j < Nj; j++)
    for (k = 0; k < Nk; k++)
// C[i][j] = C[i][j] + A[i][k]*B[j][k];
     C[i*Nj+j]=C[i*Nj+j]+A[i*Nk+k]*B[j*Nk+k];
}

void aTb_seq(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
  int i, j, k;

  for (i = 0; i < Ni; i++)
   for (j = 0; j < Nj; j++)
    for (k = 0; k < Nk; k++)
// C[i][j] = C[i][j] + A[k][i]*B[k][j];
     C[i*Nj+j]=C[i*Nj+j]+A[k*Ni+i]*B[k*Nj+j];
}

void aTbT_seq(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C, int Ni, int Nj, int Nk)
{
  int i, j, k;

  for (i = 0; i < Ni; i++)
   for (j = 0; j < Nj; j++)
    for (k = 0; k < Nk; k++)
// C[i][j] = C[i][j] + A[k][i]*B[j][k];
     C[i*Nj+j]=C[i*Nj+j]+A[k*Ni+i]*B[j*Nk+k];
}


int main(){

  float *h_A, *h_B, *h_C, *h_Cref, *d_A, *d_B, *d_C;
  int i,j,k;
  int Ni,Nj,Nk;


  printf("Specify Matrix dimension Ni, Nj, Nk: ");
  scanf("%d %d %d", &Ni,&Nj,&Nk);

  h_A = (float *) malloc(sizeof(float)*Ni*Nk);
  h_B = (float *) malloc(sizeof(float)*Nk*Nj);
  h_C = (float *) malloc(sizeof(float)*Ni*Nj);
  h_Cref = (float *) malloc(sizeof(float)*Ni*Nj);;

  for (i=0; i<Ni; i++)
   for (k=0; k<Nk; k++)
    h_A[k*Ni+i] = rand();
  for (k=0; k<Nk; k++)
   for (j=0; j<Nj; j++)
    h_B[k*Nj+j] = rand();

  
 // Allocate device memory and copy input data over to GPU
  hipMalloc(&d_A, Ni*Nk*sizeof(float));
  hipMalloc(&d_B, Nk*Nj*sizeof(float));
  hipMalloc(&d_C, Ni*Nj*sizeof(float));
  checkCUDAError("hipMalloc failure");
  hipMemcpy(d_A, h_A, Ni*Nk*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, Nk*Nj*sizeof(float), hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy H2D transfer failure");

  dim3 block(BLOCK_SIZE,BLOCK_SIZE);  
  dim3 grid(ceil(Ni/float(block.x)),ceil(Nj/float(block.y)));
  for(int version=0; version<4; version++)
  {
   for(i=0;i<Ni;i++) for(j=0;j<Nj;j++) h_Cref[i*Nj+j] = 0;
   switch (version) {
      case 0: ab_seq(h_A,h_B,h_Cref,Ni,Nj,Nk);  break;
      case 1: aTb_seq(h_A,h_B,h_Cref,Ni,Nj,Nk); break;
      case 2: abT_seq(h_A,h_B,h_Cref,Ni,Nj,Nk); break;
      case 3: aTbT_seq(h_A,h_B,h_Cref,Ni,Nj,Nk);
    }
    for(int trial=0;trial<3;trial++)
    {
     for(i=0;i<Ni;i++) for(j=0;j<Nj;j++) h_C[i*Nj+j] = 0; 
      printf("Trial %d: ",trial);
      hipEventCreate(&start);
      hipEventCreate(&stop);
      hipEventRecord(start);
      // Launch kernel
      switch (version) {
      case 0: if(Ni%BLOCK_SIZE == 0 && Nj%BLOCK_SIZE==0 && Nk%BLOCK_SIZE==0)
              {
                if(Ni == Nj && Nj==Nk)
                {
                  dim3 grid(ceil((Ni/4)/float(block.x)),ceil((Nj/4)/float(block.y)));
                  ab_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("AB "); 
                }
                else if(Ni==Nj && Nj > Nk)
                {
                  dim3 grid(ceil((Ni/4)/float(block.x)),ceil((Nj/4)/float(block.y)));
                  ab_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("AB "); 
                }
                else
                {
                  dim3 grid(ceil(Ni/float(block.x)),ceil(Nj/float(block.y)));
                  ab_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("AB "); 
                }
              }
              else
              {
                dim3 grid(ceil(Ni/float(block.x)),ceil(Nj/float(block.y)));
                ab_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("AB "); 
                // dim3 grid(ceil(Ni/float(block.x)),ceil(Nj/float(block.y)));
                // ab_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni-(Ni%(BLOCK_SIZE*4)),Nj-(Nj%(BLOCK_SIZE*4)),Nk-(Nk%(BLOCK_SIZE*4)));

                // dim3 grid(ceil(Ni/float(block.x)),ceil(Nj/float(block.y)));
                // ab_gpu_vertical<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk,Ni-(Ni%(BLOCK_SIZE*4)),Nj%(BLOCK_SIZE*4));  

                // dim3 grid(ceil(Ni/float(block.x)),ceil(Nj/float(block.y)));
                // ab_gpu_horizontal<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk,Ni%(BLOCK_SIZE*4),Nj-Nj%(BLOCK_SIZE*4));  
                
                // dim3 grid(ceil(Ni/float(block.x)),ceil(Nj/float(block.y)));
                // ab_gpu_small_square<<<grid, block>>>(d_A, d_B, d_C,Ni%(BLOCK_SIZE*4),Nj%(BLOCK_SIZE*4),Nj%(BLOCK_SIZE*4));
                // printf("AB "); 
              }
              break;
      case 1: if(Ni%BLOCK_SIZE == 0 && Nj%BLOCK_SIZE==0 && Nk%BLOCK_SIZE==0)
              {
                if((Ni == Nj && Nj==Nk) )
                {
                  dim3 grid(ceil((Ni/4)/float(block.x)),ceil((Nj/4)/float(block.y)));
                  aTb_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("ATB ");
                }
                else
                {
                  dim3 grid(ceil(Ni/float(block.x)),ceil(Nj/float(block.y)));
                  aTb_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("ATB ");
                }
              }
              else
              {
                dim3 grid(ceil(Ni/float(block.x)),ceil(Nj/float(block.y)));
                aTb_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("ATB ");
              }
              break;
      case 2: if(Ni%BLOCK_SIZE == 0 && Nj%BLOCK_SIZE==0 && Nk%BLOCK_SIZE==0)
              {
                if((Ni == Nj && Nj==Nk) || (Ni==Nj && Nj>Nk))
                {
                  dim3 grid(ceil((Ni/4)/float(block.x)),ceil((Nj/4)/float(block.y)));
                  abT_gpu<<<grid, block>>>(d_A, d_B, d_C, Ni, Nj, Nk); printf("ABT ");
                }
                else
                {
                  dim3 grid(ceil(Ni/float(block.x)),ceil(Nj/float(block.y)));
                  abT_gpu<<<grid, block>>>(d_A, d_B, d_C, Ni, Nj, Nk); printf("ABT ");
                }
              }
              else
              {
                dim3 grid(ceil(Ni/float(block.x)),ceil(Nj/float(block.y)));
                abT_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("ABT ");
              }
              break;
      case 3: if(Ni%BLOCK_SIZE == 0 && Nj%BLOCK_SIZE==0 && Nk%BLOCK_SIZE==0)
              {
                if((Ni == Nj && Nj==Nk) || (Ni==Nj && Nj>Nk))
                {
                  dim3 grid(ceil((Ni/4)/float(block.x)),ceil((Nj/4)/float(block.y)));
                  aTbT_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("ATBT ");
                }
                else
                {
                  dim3 grid(ceil(Ni/float(block.x)),ceil(Nj/float(block.y)));
                  aTbT_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("ATBT ");
                }
              }
              else
              {
                dim3 grid(ceil(Ni/float(block.x)),ceil(Nj/float(block.y)));
                aTbT_gpu<<<grid, block>>>(d_A, d_B, d_C,Ni,Nj,Nk); printf("ATBT ");
              }
              break;
      }
      checkCUDAError("GPU kernel launch failure");
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&elapsedTime, start,stop);
      hipDeviceSynchronize();
      // Copy results back to host
      hipMemcpy(h_C, d_C, Ni*Nj*sizeof(float), hipMemcpyDeviceToHost);
      checkCUDAError("hipMemcpy D2H");
      for (int i = 0; i < Ni*Nj; i++) if (fabs((h_C[i]-h_Cref[i])/h_Cref[i])>threshold) {printf("Error: mismatch at linearized index %d, was: %f, should be: %f\n", i, h_C[i], h_Cref[i]); return -1;}
      printf("GFLOPS: %.2f\n",2.0e-6*Ni*Nj*Nk/elapsedTime);
     }
  }
  return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

